#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "windows.h"
#include ""
typedef unsigned char uchar;

float* ReadBMP(const char *bmpName, int *width, int *height)
{
	FILE *fp;
	uchar *img_raw; float *image;
	int bmpwidth, bmpheight, linebyte, npixels, i, j; if ((fp = fopen(bmpName, "rb")) == NULL)
	{
		printf("Failed to open the image.\n");
		return 0;
	} if (fseek(fp, sizeof(BITMAPFILEHEADER), 0))
	{
		printf("Failed to skip the file header.\n");
		return 0;
	} BITMAPINFOHEADER bmpInfoHeader;
	fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	bmpwidth = bmpInfoHeader.biWidth;
	bmpheight = bmpInfoHeader.biHeight;
	npixels = bmpwidth*bmpheight;
	linebyte = (bmpwidth * 24 / 8 + 3) / 4 * 4; img_raw = (uchar*)malloc(linebyte*bmpheight);
	fread(img_raw, linebyte*bmpheight, 1, fp); image = (float*)malloc(sizeof(float)*npixels);
	for (i = 0; i < bmpheight; i++)
		for (j = 0; j < bmpwidth; j++)
			image[i*bmpwidth + j] = (float)img_raw[i*linebyte + j * 3];
	*width = bmpwidth;
	*height = bmpheight; free(img_raw);
	fclose(fp);
	return image;
}

void MarkAndSave(const char* bmpName, int X1, int Y1, int X2, int Y2, const char* outputBmpName)
{
	FILE *fp;
	uchar *img_raw; float *image;
	BITMAPFILEHEADER bmpFileHeader;
	BITMAPINFOHEADER bmpInfoHeader;
	int bmpwidth, bmpheight, linebyte, npixels;
	if ((fp = fopen(bmpName, "rb")) == NULL)
	{
		printf("Failed to open the original image.\n");
		return;
	} fread(&bmpFileHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	bmpwidth = bmpInfoHeader.biWidth;
	bmpheight = bmpInfoHeader.biHeight;
	npixels = bmpwidth*bmpheight;
	linebyte = (bmpwidth * 24 / 8 + 3) / 4 * 4; img_raw = (uchar*)malloc(linebyte*bmpheight);
	fread(img_raw, linebyte*bmpheight, 1, fp);
	fclose(fp); if (X1 < 0 || X2 >= bmpwidth || Y1 < 0 || Y2 >= bmpheight)
	{
		printf("Invalid rectangle position!\n");
		return;
	}
	int i;
	for (i = X1; i <= X2; i++)
	{
		img_raw[Y1*linebyte + i * 3] = 0;
		img_raw[Y1*linebyte + i * 3 + 1] = 0;
		img_raw[Y1*linebyte + i * 3 + 2] = 255;
		img_raw[Y2*linebyte + i * 3] = 0;
		img_raw[Y2*linebyte + i * 3 + 1] = 0;
		img_raw[Y2*linebyte + i * 3 + 2] = 255;
	}
	for (i = Y1 + 1; i < Y2; i++)
	{
		img_raw[i*linebyte + X1 * 3] = 0;
		img_raw[i*linebyte + X1 * 3 + 1] = 0;
		img_raw[i*linebyte + X1 * 3 + 2] = 255;
		img_raw[i*linebyte + X2 * 3] = 0;
		img_raw[i*linebyte + X2 * 3 + 1] = 0;
		img_raw[i*linebyte + X2 * 3 + 2] = 255;
	} if ((fp = fopen(outputBmpName, "wb")) == NULL)
	{
		printf("Failed to open the output image.\n");
		return;
	}
	fwrite(&bmpFileHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	fwrite(img_raw, linebyte*bmpheight, 1, fp); free(img_raw);
	fclose(fp);
}

//use to get the Template's data
float* getTemdata(float *tem, float *output, int K) {
	for (int i = 0; i < K*K; i++) {
		output[0] += tem[i] / (K*K);
	}
	for (int i = 0; i < K*K; i++) {
		output[1] += tem[i] * tem[i] / (K*K);
	}
	output[1] = output[1] - output[0] * output[0];

	for (int i = 0; i < K*K; i++) {
		int j = i % K;
		int j_minus_x = j - K / 2;
		output[2] += tem[i] * j_minus_x;
	}
	output[2] = 4 * output[2] / (K*K*K);

	for (int i = 0; i < K*K; i++) {
		int l = i / K;
		int l_minus_y = l - K / 2;
		output[3] += tem[i] * l_minus_y;
	}
	output[3] = 4 * output[3] / (K*K*K); return output;
}

//TO DO: WRITE KERNELS HERE
__global__ void Row_Cumulate_gpu(float *Image, float *L1_dev_row, float *L2_dev_row, float *L3_dev_row, float *L4_dev_row, int m, int n)
{
	int index = blockIdx.x * 128 + threadIdx.x;
	int i, j, k;

	if (index >= n) return;

	L1_dev_row[index * m] = Image[index * m];
	for (j = 1; j < m; j++)
	{
		L1_dev_row[index * m + j] = L1_dev_row[index * m + j - 1] + Image[index * m + j];
	}

	L2_dev_row[index * m] = Image[index * m] * Image[index * m];
	for (j = 1; j < m; j++)
	{
		L2_dev_row[index * m + j] = L2_dev_row[index * m + j - 1] + Image[index * m + j] * Image[index * m + j];
	}

	L3_dev_row[index * m] = Image[index * m] * 0;
	for (j = 1; j < m; j++)
	{
		L3_dev_row[index * m + j] = L3_dev_row[index * m + j - 1] + Image[index * m + j] * (j);
	}

	L4_dev_row[index * m] = Image[index * m] * (index);
	for (j = 1; j < m; j++)
	{
		L4_dev_row[index * m + j] = L4_dev_row[index * m + j - 1] + Image[index * m + j] * (index);
	}
}

__global__ void Column_Cumulate_gpu(float *L1_Row, float *L2_Row, float *L3_Row, float *L4_Row, float *L1, float *L2, float *L3, float *L4, int m, int n)
{
	int index = blockIdx.x * 128 + threadIdx.x;
	int i, j;

	if (index >= m) return;

	L1[index] = L1_Row[index];
	for (j = 1; j < n; j++)
	{
		L1[index + j * m] = L1[index + (j - 1) * m] + L1_Row[index + j * m];
	}

	L2[index] = L2_Row[index];
	for (j = 1; j < n; j++)
	{
		L2[index + j * m] = L2[index + (j - 1) * m] + L2_Row[index + j * m];
	}

	L3[index] = L3_Row[index];
	for (j = 1; j < n; j++)
	{
		L3[index + j * m] = L3[index + (j - 1) * m] + L3_Row[index + j * m];
	}

	L4[index] = L4_Row[index];
	for (j = 1; j < n; j++)
	{
		L4[index + j * m] = L4[index + (j - 1) * m] + L4_Row[index + j * m];
	}
}

__global__ void kernel_3(float *L1_dev, float *L2_dev, float *L3_dev, float *L4_dev, float *output_dev, int N, int M, int L, float *tem_inf)
{
	int row_ID, col_ID;
	float S1, S2, S3, S4, V1, V2, V3, V4;
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	row_ID = index / M;
	col_ID = index % M;

	if (col_ID < (L / 2) || (col_ID >(M - L / 2 - (L % 2 > 0))) || (row_ID > (N - L / 2 - (L % 2 > 0))) || (row_ID < (L / 2))) {
		V1 = 0;
		V2 = 0;
		V3 = 0;
		V4 = 0;
	}

	else {
		if ((col_ID != L / 2) && (row_ID != L / 2)) {
			S1 = L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1] - L1_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] + L1_dev[(row_ID - L / 2 - 1)*M + (col_ID - L / 2 - 1)];
			S2 = L2_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L2_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1] - L2_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] + L2_dev[(row_ID - L / 2 - 1)*M + (col_ID - L / 2 - 1)];
			S3 = L3_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L3_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1] - L3_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] + L3_dev[(row_ID - L / 2 - 1)*M + (col_ID - L / 2 - 1)];
			S4 = L4_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L4_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1] - L4_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] + L4_dev[(row_ID - L / 2 - 1)*M + (col_ID - L / 2 - 1)];
			V1 = S1 / (L*L);
			V2 = S2 / (L*L) - (V1*V1);
			V3 = 4 * (S3 - col_ID*S1) / (L*L*L);
			V4 = 4 * (S4 - row_ID*S1) / (L*L*L);
		}
		else if ((col_ID == L / 2) && (row_ID != L / 2)) {
			S1 = L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L1_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			S2 = L2_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L2_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			S3 = L3_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L3_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			S4 = L4_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L4_dev[(row_ID - L / 2 - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			V1 = S1 / (L*L);
			V2 = S2 / (L*L) - (V1*V1);
			V3 = 4 * (S3 - col_ID*S1) / (L*L*L);
			V4 = 4 * (S4 - row_ID*S1) / (L*L*L);
		}
		else if ((col_ID != L / 2) && (row_ID == L / 2)) {
			S1 = L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1];
			S2 = L2_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1];
			S3 = L3_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1];
			S4 = L4_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1] - L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID - L / 2 - 1];
			V1 = S1 / (L*L);
			V2 = S2 / (L*L) - (V1*V1);
			V3 = 4 * (S3 - col_ID*S1) / (L*L*L);
			V4 = 4 * (S4 - row_ID*S1) / (L*L*L);
		}
		else {
			S1 = L1_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			S2 = L2_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			S3 = L3_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			S4 = L4_dev[(row_ID + L / 2 + (L % 2 > 0) - 1)*M + col_ID + L / 2 + (L % 2 > 0) - 1];
			V1 = S1 / (L*L);
			V2 = S2 / (L*L) - (V1*V1);
			V3 = 4 * (S3 - col_ID*S1) / (L*L*L);
			V4 = 4 * (S4 - row_ID*S1) / (L*L*L);
		}

	}
	output_dev[index] = (V1 - tem_inf[0])*(V1 - tem_inf[0]) + (V2 - tem_inf[1])*(V2 - tem_inf[1]) + (V3 - tem_inf[2])*(V3 - tem_inf[2]) + (V4 - tem_inf[3])*(V4 - tem_inf[3]);
}

int main()
{
	//Just an example here - you are free to modify them
	int I_width, I_height, T_width, T_height;
	float *I, *T; float *L1, *L2, *L3, *L4, *output;
	float *L1_dev, *L2_dev, *L3_dev, *L4_dev, *output_dev, *tem_inf_dev;
	float *L1_row, *L2_row, *L3_row, *L4_row;
	float *L1_dev_row, *L2_dev_row, *L3_dev_row, *L4_dev_row;
	float *I_dev;
	float x1, y1, x2, y2;
	float time_kernel1, time_kernel2, time_kernel3;

	char I_path[] = "lena.bmp";
	char T_path[] = "lena_t.bmp";
	char out_path[] = "output.bmp";
	I = ReadBMP(I_path, &I_width, &I_height);
	T = ReadBMP(T_path, &T_width, &T_height);

	int N = I_height, M = I_width, K = T_width;
	int blocksize = 512;
	int blocknumber0 = N / 128 + (N % 128>0);
	int blocknumber1 = M / 128 + (M % 128>0);
	int blocknumber2 = M*N / blocksize + (((M*N) % blocksize)> 0);

	size_t memsize_of_L = M*N * sizeof(float);

	L1_row = (float *)malloc(memsize_of_L);
	L2_row = (float *)malloc(memsize_of_L);
	L3_row = (float *)malloc(memsize_of_L);
	L4_row = (float *)malloc(memsize_of_L);
	L1 = (float *)malloc(memsize_of_L);
	L2 = (float *)malloc(memsize_of_L);
	L3 = (float *)malloc(memsize_of_L);
	L4 = (float *)malloc(memsize_of_L);
	I = (float *)malloc(memsize_of_L);
	T = (float *)malloc(K*K * sizeof(float));

	output = (float *)malloc(memsize_of_L);
	hipMalloc((void **)&L1_dev_row, memsize_of_L);
	hipMalloc((void **)&L2_dev_row, memsize_of_L);
	hipMalloc((void **)&L3_dev_row, memsize_of_L);
	hipMalloc((void **)&L4_dev_row, memsize_of_L);
	hipMalloc((void **)&L1_dev, memsize_of_L);
	hipMalloc((void **)&L2_dev, memsize_of_L);
	hipMalloc((void **)&L3_dev, memsize_of_L);
	hipMalloc((void **)&L4_dev, memsize_of_L);
	hipMalloc((void **)&I_dev, memsize_of_L);
	hipMalloc((void **)&output_dev, memsize_of_L);
	hipMalloc((void **)&tem_inf_dev, 4 * sizeof(float));

	I = ReadBMP(I_path, &I_width, &I_height);
	T = ReadBMP(T_path, &T_width, &T_height);
	
	float tem_inf[4] = { 0,0,0,0 };
	float *Tem_data = getTemdata(T, tem_inf, K);

	hipMemcpy(I_dev, I, memsize_of_L, hipMemcpyHostToDevice);
	//kernel1 and kernel2
	hipEvent_t start_kernel1, end_kernel1;
	hipEventCreate(&start_kernel1);
	hipEventCreate(&end_kernel1);
	hipEventRecord(start_kernel1, 0);
	Row_Cumulate_gpu << <blocknumber0, 128 >> > (I_dev, L1_dev_row, L2_dev_row, L3_dev_row, L4_dev_row, M, N);
	hipEventRecord(end_kernel1, 0);
	hipEventSynchronize(start_kernel1);
	hipEventSynchronize(end_kernel1);
	hipEventElapsedTime(&time_kernel1, start_kernel1, end_kernel1);

	hipEvent_t start_kernel2, end_kernel2;
	hipEventCreate(&start_kernel2);
	hipEventCreate(&end_kernel2);
	hipEventRecord(start_kernel2, 0);
	Column_Cumulate_gpu << < blocknumber1, 128 >> > (L1_dev_row, L2_dev_row, L3_dev_row, L4_dev_row, L1_dev, L2_dev, L3_dev, L4_dev, M, N);
	hipEventRecord(end_kernel2, 0);
	hipEventSynchronize(start_kernel2);
	hipEventSynchronize(end_kernel2);
	hipEventElapsedTime(&time_kernel2, start_kernel2, end_kernel2);

	hipMemcpy(tem_inf_dev, Tem_data, 4 * sizeof(float), hipMemcpyHostToDevice);
	//kernel3
	kernel_3 << <blocknumber2, blocksize >> > (L1_dev, L2_dev, L3_dev, L4_dev, output_dev, N, M, K, tem_inf_dev);
	hipEvent_t start_kernel3, end_kernel3;
	hipEventCreate(&start_kernel3);
	hipEventCreate(&end_kernel3);
	hipEventRecord(start_kernel3, 0);
	kernel_3 << <blocknumber2, blocksize >> > (L1_dev, L2_dev, L3_dev, L4_dev, output_dev, N, M, K, tem_inf_dev);
	hipEventRecord(end_kernel3, 0);
	hipEventSynchronize(start_kernel3);
	hipEventSynchronize(end_kernel3);
	hipEventElapsedTime(&time_kernel3, start_kernel3, end_kernel3);

	hipMemcpy(output, output_dev, memsize_of_L, hipMemcpyDeviceToHost);

	int pos = 0;
	float min = 100;
	for (int i = 0; i < (M *N); i++) {
		if (output[i] < min) {
			min = output[i];
			pos = i;
		}
	}

	float row_pos = pos / M;
	float col_pos = pos % M;
	y1 = row_pos - K / 2;
	x1 = col_pos - K / 2;
	y2 = row_pos + K / 2;
	x2 = col_pos + K / 2;

	printf("Success match.\n");
	printf("Average GPU running time (kernel1): %f ms\n", time_kernel1);
	printf("Average GPU running time (kernel2): %f ms\n", time_kernel2);
	printf("Average GPU running time (kernel3): %f ms\n", time_kernel3);
	MarkAndSave(I_path, x1, y1, x2, y2, out_path);
	free(I); free(T);
	free(L1_row); free(L2_row); free(L3_row); free(L4_row);
	free(L1); free(L2); free(L3); free(L4);
	free(output);
	hipFree(L1_dev_row); hipFree(L2_dev_row); hipFree(L3_dev_row); hipFree(L4_dev_row);
	hipFree(L1_dev); hipFree(L2_dev); hipFree(L3_dev); hipFree(L4_dev);
	hipFree(tem_inf_dev); hipFree(I_dev); hipFree(output_dev);
	return 0;
}
